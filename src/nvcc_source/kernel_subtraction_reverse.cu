#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_subtraction_reverse_RScalar(
  RScalar *dev_a,
  const RScalar *dev_b,
  RScalar coef,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_a[tid] += coef * dev_b[tid];
}

extern "C" void launch_subtraction_reverse_RScalar(
  Stream stream,
  RScalar* a, 
  const RScalar* b, 
  const RScalar coef,
  len_t N
) {
  __kernel_subtraction_reverse_RScalar<<<1, GRID_1D(N), 32, getStream(stream)>>>(a, b, coef, N);
}

__global__ void __kernel_subtraction_reverse_CScalar(
  CScalar *dev_a,
  const CScalar *dev_b,
  const RScalar coef,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {
    dev_a[tid].r += coef * dev_b[tid].r;
    dev_a[tid].i += coef * dev_b[tid].i;
  }
}

extern "C" void launch_subtraction_reverse_CScalar(
  Stream stream,
  CScalar* a, 
  const CScalar* b, 
  const RScalar coef,
  len_t N
) {
  __kernel_subtraction_reverse_CScalar<<<1, GRID_1D(N), 32, getStream(stream)>>>(a, b, coef, N);
  CUDA_ASSERT(hipDeviceSynchronize());
}

