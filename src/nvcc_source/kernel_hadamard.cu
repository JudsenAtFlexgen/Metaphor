#include "hip/hip_runtime.h"
#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard
__global__ void __kernel_hadamard_RScalar(
  const RScalar *dev_a,
  const RScalar *dev_b,
  RScalar *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_c[tid] = dev_a[tid] * dev_b[tid];
}

extern "C" void launch_hadamard_RScalar(
  const RScalar* a,
  const RScalar* b, 
  RScalar* c, 
  len_t N
) {
  __kernel_hadamard_RScalar<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_CScalar(
  const CScalar *dev_a,
  const CScalar *dev_b,
  CScalar *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    dev_c[tid].r = (dev_a[tid].r * dev_b[tid].r - dev_a[tid].i * dev_b[tid].i);
    dev_c[tid].i = (dev_a[tid].r * dev_b[tid].i + dev_a[tid].i * dev_b[tid].r);
  }
}

extern "C" void launch_hadamard_CScalar(
  const CScalar* a,
  const CScalar* b, 
  CScalar* c, 
  len_t N
) {
  __kernel_hadamard_CScalar<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

