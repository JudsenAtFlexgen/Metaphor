#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_tanh_RScalar(
  const RScalar *a_value,
        RScalar *b_value,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    b_value[tid] = rtanh(a_value[tid]);
  }
}

extern "C" void launch_tanh_RScalar(
  Stream stream,
  const RScalar* a,
        RScalar* b, 
  len_t N
) {
  __kernel_tanh_RScalar<<<1, GRID_1D(N), 32, getStream(stream)>>>(a, b, N);
}
