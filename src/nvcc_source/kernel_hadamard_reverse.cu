#include "hip/hip_runtime.h"
#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard_reverse
__global__ void __kernel_hadamard_reverse_RScalar(
  RScalar *grads_a,
  const RScalar *value_b,
  const RScalar *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    grads_a[tid] += value_b[tid] * grads_c[tid];
}

extern "C" void launch_hadamard_reverse_RScalar(
  RScalar *grads_a,
  const RScalar *value_b,
  const RScalar *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_RScalar<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_reverse_CScalar(
  CScalar *grads_a,
  const CScalar *value_b,
  const CScalar *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    grads_a[tid].r = (value_b[tid].r * grads_c[tid].r - value_b[tid].i * grads_c[tid].i);
    grads_a[tid].i = (value_b[tid].r * grads_c[tid].i + value_b[tid].i * grads_c[tid].r);
  }
}

extern "C" void launch_hadamard_reverse_CScalar(
  CScalar *grads_a,
  const CScalar *value_b,
  const CScalar *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_CScalar<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

