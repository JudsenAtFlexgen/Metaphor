#include "hip/hip_runtime.h"
#include "../kernel_header.h"

// CUDA Kernel for Vector subtraction
__global__ void __kernel_subtraction_RScalar(
  const RScalar *dev_a,
  const RScalar *dev_b,
  RScalar *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_c[tid] = dev_a[tid] - dev_b[tid];
}

extern "C" void launch_subtraction_RScalar(
  Stream stream,
  const RScalar* a,
  const RScalar* b, 
  RScalar* c, 
  len_t N
) {
  __kernel_subtraction_RScalar<<<GRID_1D(N), dim3(32), 0, getStream(stream)>>>(a, b, c, N);
}

__global__ void __kernel_subtraction_CScalar(
  const CScalar *dev_a,
  const CScalar *dev_b,
  CScalar *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {
    dev_c[tid].r = dev_a[tid].r - dev_b[tid].r;
    dev_c[tid].i = dev_a[tid].i - dev_b[tid].i;
  }
}

extern "C" void launch_subtraction_CScalar(
  Stream stream,
  const CScalar* a,
  const CScalar* b, 
  CScalar* c, 
  len_t N
) {
  __kernel_subtraction_CScalar<<<GRID_1D(N), dim3(32), 0, getStream(stream)>>>(a, b, c, N);
}
