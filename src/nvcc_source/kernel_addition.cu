#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_addition_RScalar(
  const RScalar *dev_a,
  const RScalar *dev_b,
  RScalar *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_c[tid] = dev_a[tid] + dev_b[tid];
}

extern "C" void launch_addition_RScalar(
  const RScalar* a,
  const RScalar* b, 
  RScalar* c, 
  len_t N
) {
  __kernel_addition_RScalar<<<GRID_1D(N), 32>>>(a, b, c, N);
  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_addition_CScalar(
  const CScalar *dev_a,
  const CScalar *dev_b,
  CScalar *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {
    dev_c[tid].r = dev_a[tid].r + dev_b[tid].r;
    dev_c[tid].i = dev_a[tid].i + dev_b[tid].i;
  }
}

extern "C" void launch_addition_CScalar(
  const CScalar* a,
  const CScalar* b, 
  CScalar* c, 
  len_t N
) {
  __kernel_addition_CScalar<<<GRID_1D(N), 32>>>(a, b, c, N);
  CUDA_ASSERT(hipDeviceSynchronize());
}

