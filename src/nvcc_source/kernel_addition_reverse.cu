#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_addition_reverse_RScalar(
  RScalar *dev_a,
  const RScalar *dev_b,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_a[tid] += dev_b[tid];
}

extern "C" void launch_addition_reverse_RScalar(
  Stream stream,
  RScalar* a, 
  const RScalar* b, 
  len_t N
) {
  __kernel_addition_reverse_RScalar<<<1, GRID_1D(N), 32, getStream(stream)>>>(a, b, N);
}

__global__ void __kernel_addition_reverse_CScalar(
  CScalar *dev_a,
  const CScalar *dev_b,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {
    dev_a[tid].r += dev_b[tid].r;
    dev_a[tid].i += dev_b[tid].i;
  }
}

extern "C" void launch_addition_reverse_CScalar(
  Stream stream,
  CScalar* a, 
  const CScalar* b, 
  len_t N
) {
  __kernel_addition_reverse_CScalar<<<1, GRID_1D(N), 32, getStream(stream)>>>(a, b, N);
}
