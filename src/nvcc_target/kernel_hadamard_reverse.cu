#include "hip/hip_runtime.h"
#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard_reverse
__global__ void __kernel_hadamard_reverse_r16(
  r16 *grads_a,
  const r16 *value_b,
  const r16 *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    grads_a[tid] += value_b[tid] * grads_c[tid];
}

extern "C" void launch_hadamard_reverse_r16(
  r16 *grads_a,
  const r16 *value_b,
  const r16 *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_r16<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_reverse_c16(
  c16 *grads_a,
  const c16 *value_b,
  const c16 *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    grads_a[tid].r = (value_b[tid].r * grads_c[tid].r - value_b[tid].i * grads_c[tid].i);
    grads_a[tid].i = (value_b[tid].r * grads_c[tid].i + value_b[tid].i * grads_c[tid].r);
  }
}

extern "C" void launch_hadamard_reverse_c16(
  c16 *grads_a,
  const c16 *value_b,
  const c16 *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_c16<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard_reverse
__global__ void __kernel_hadamard_reverse_r32(
  r32 *grads_a,
  const r32 *value_b,
  const r32 *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    grads_a[tid] += value_b[tid] * grads_c[tid];
}

extern "C" void launch_hadamard_reverse_r32(
  r32 *grads_a,
  const r32 *value_b,
  const r32 *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_r32<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_reverse_c32(
  c32 *grads_a,
  const c32 *value_b,
  const c32 *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    grads_a[tid].r = (value_b[tid].r * grads_c[tid].r - value_b[tid].i * grads_c[tid].i);
    grads_a[tid].i = (value_b[tid].r * grads_c[tid].i + value_b[tid].i * grads_c[tid].r);
  }
}

extern "C" void launch_hadamard_reverse_c32(
  c32 *grads_a,
  const c32 *value_b,
  const c32 *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_c32<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard_reverse
__global__ void __kernel_hadamard_reverse_r64(
  r64 *grads_a,
  const r64 *value_b,
  const r64 *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    grads_a[tid] += value_b[tid] * grads_c[tid];
}

extern "C" void launch_hadamard_reverse_r64(
  r64 *grads_a,
  const r64 *value_b,
  const r64 *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_r64<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_reverse_c64(
  c64 *grads_a,
  const c64 *value_b,
  const c64 *grads_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    grads_a[tid].r = (value_b[tid].r * grads_c[tid].r - value_b[tid].i * grads_c[tid].i);
    grads_a[tid].i = (value_b[tid].r * grads_c[tid].i + value_b[tid].i * grads_c[tid].r);
  }
}

extern "C" void launch_hadamard_reverse_c64(
  c64 *grads_a,
  const c64 *value_b,
  const c64 *grads_c,
  len_t N
) {
  __kernel_hadamard_reverse_c64<<<GRID_1D(N), 32>>>(grads_a, value_b, grads_c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

