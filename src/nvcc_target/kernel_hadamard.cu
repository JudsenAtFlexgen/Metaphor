#include "hip/hip_runtime.h"
#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard
__global__ void __kernel_hadamard_r16(
  const r16 *dev_a,
  const r16 *dev_b,
  r16 *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_c[tid] = dev_a[tid] * dev_b[tid];
}

extern "C" void launch_hadamard_r16(
  const r16* a,
  const r16* b, 
  r16* c, 
  len_t N
) {
  __kernel_hadamard_r16<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_c16(
  const c16 *dev_a,
  const c16 *dev_b,
  c16 *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    dev_c[tid].r = (dev_a[tid].r * dev_b[tid].r - dev_a[tid].i * dev_b[tid].i);
    dev_c[tid].i = (dev_a[tid].r * dev_b[tid].i + dev_a[tid].i * dev_b[tid].r);
  }
}

extern "C" void launch_hadamard_c16(
  const c16* a,
  const c16* b, 
  c16* c, 
  len_t N
) {
  __kernel_hadamard_c16<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard
__global__ void __kernel_hadamard_r32(
  const r32 *dev_a,
  const r32 *dev_b,
  r32 *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_c[tid] = dev_a[tid] * dev_b[tid];
}

extern "C" void launch_hadamard_r32(
  const r32* a,
  const r32* b, 
  r32* c, 
  len_t N
) {
  __kernel_hadamard_r32<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_c32(
  const c32 *dev_a,
  const c32 *dev_b,
  c32 *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    dev_c[tid].r = (dev_a[tid].r * dev_b[tid].r - dev_a[tid].i * dev_b[tid].i);
    dev_c[tid].i = (dev_a[tid].r * dev_b[tid].i + dev_a[tid].i * dev_b[tid].r);
  }
}

extern "C" void launch_hadamard_c32(
  const c32* a,
  const c32* b, 
  c32* c, 
  len_t N
) {
  __kernel_hadamard_c32<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

#include "../kernel_header.h"

// CUDA Kernel for Vector hadamard
__global__ void __kernel_hadamard_r64(
  const r64 *dev_a,
  const r64 *dev_b,
  r64 *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_c[tid] = dev_a[tid] * dev_b[tid];
}

extern "C" void launch_hadamard_r64(
  const r64* a,
  const r64* b, 
  r64* c, 
  len_t N
) {
  __kernel_hadamard_r64<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

__global__ void __kernel_hadamard_c64(
  const c64 *dev_a,
  const c64 *dev_b,
  c64 *dev_c,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {    
    dev_c[tid].r = (dev_a[tid].r * dev_b[tid].r - dev_a[tid].i * dev_b[tid].i);
    dev_c[tid].i = (dev_a[tid].r * dev_b[tid].i + dev_a[tid].i * dev_b[tid].r);
  }
}

extern "C" void launch_hadamard_c64(
  const c64* a,
  const c64* b, 
  c64* c, 
  len_t N
) {
  __kernel_hadamard_c64<<<GRID_1D(N), 32>>>(a, b, c, N);

  CUDA_ASSERT(hipDeviceSynchronize());
}

