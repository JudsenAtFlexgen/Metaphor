
#include "device_utils.h"

#include "kernel_header.h"

extern "C" void mpMemAlloc(void** ptr, size_t N) {
  CUDA_ASSERT(hipMalloc(ptr, N));
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" void mpMemcpyHtoD(void* dev_ptr, const void* cpu_ptr, size_t bytecount) {
  CUDA_ASSERT(hipMemcpy(dev_ptr, cpu_ptr, bytecount, hipMemcpyHostToDevice));
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" void mpMemcpyDtoH(void* cpu_ptr, void const* dev_ptr, size_t bytecount) {
  CUDA_ASSERT(hipMemcpy(cpu_ptr, dev_ptr, bytecount, hipMemcpyDeviceToHost));
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" void mpMemFree(void* dev_ptr) {
  CUDA_ASSERT(hipFree(dev_ptr));
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" void mpDeviceSynchronize() {
  CUDA_ASSERT(hipDeviceSynchronize());
}

