
//#include "device_utils.h"

#include "kernel_header.h"

extern "C" void* mpMemAlloc(size_t N, void* stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr;
  CURESULT_ASSERT(hipMallocAsync(&dptr, N, _stream));
  return (void*)dptr;
}
extern "C" void mpMemcpyHtoD(void* dev_ptr, const void* cpu_ptr, size_t N, void* stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyHtoDAsync(dptr, cpu_ptr, N, _stream));
}
extern "C" void mpMemcpyDtoH(void* cpu_ptr, void const* dev_ptr, size_t N, void* stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyDtoHAsync(cpu_ptr, dptr, N, _stream));
}
extern "C" void mpMemFree(void* dev_ptr, void* stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipFreeAsync(dptr, _stream));
}
extern "C" void mpStreamSynchronize(void* stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  CURESULT_ASSERT(hipStreamSynchronize(_stream));
}
extern "C" void mpDeviceSynchronize() {
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" void* mpInitStream() {
  hipStream_t stream = nullptr;
  CURESULT_ASSERT(hipStreamCreateWithFlags(&stream, hipStreamDefault));
  return reinterpret_cast<void*>(stream);
}
extern "C" void mpDeinitStream(void* stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  CURESULT_ASSERT(hipStreamDestroy(_stream));
}
extern "C" void initDevice(unsigned device_number) {

    CURESULT_ASSERT(hipInit(0));

    hipDevice_t device;
    hipCtx_t context;
    int device_count = 0;

    CURESULT_ASSERT(hipGetDeviceCount(&device_count));

    if (device_count <= device_number) {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    CURESULT_ASSERT(hipDeviceGet(&device, device_number));

    CURESULT_ASSERT(hipCtxCreate(&context, 0, device));
}
