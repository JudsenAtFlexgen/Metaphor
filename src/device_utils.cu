
#include "device_utils.h"

extern "C" void* mpMemAlloc(size_t N, Stream stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr;
  CURESULT_ASSERT(hipMallocAsync(&dptr, N, _stream));
  return (void*)dptr;
}
extern "C" void mpMemcpyHtoD(void* dev_ptr, const void* cpu_ptr, size_t N, Stream stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyHtoDAsync(dptr, cpu_ptr, N, _stream));
}
extern "C" void mpMemcpyDtoH(void* cpu_ptr, void const* dev_ptr, size_t N, Stream stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyDtoHAsync(cpu_ptr, dptr, N, _stream));
}
extern "C" void mpMemFree(void* dev_ptr, Stream stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipFreeAsync(dptr, _stream));
}
extern "C" void mpStreamSynchronize(Stream stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  CURESULT_ASSERT(hipStreamSynchronize(_stream));
}
extern "C" void mpDeviceSynchronize() {
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" Stream mpInitStream() {
  hipStream_t stream_ptr = nullptr;
  CURESULT_ASSERT(hipStreamCreateWithFlags(&stream_ptr, hipStreamDefault));
  return Stream { .ptr = reinterpret_cast<void*>(stream_ptr) };
}
extern "C" void mpDeinitStream(Stream stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  CURESULT_ASSERT(hipStreamDestroy(_stream));
}
extern "C" void initDevice(unsigned device_number) {

    CURESULT_ASSERT(hipInit(0));

    hipDevice_t device;
    hipCtx_t context;
    int device_count = 0;

    CURESULT_ASSERT(hipGetDeviceCount(&device_count));

    if (device_count <= device_number) {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    CURESULT_ASSERT(hipDeviceGet(&device, device_number));

    CURESULT_ASSERT(hipCtxCreate(&context, 0, device));
}
