
#include "device_utils.h"

extern "C" void* mpMemAlloc(len_t N, StreamCtx stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr;
  CURESULT_ASSERT(hipMallocAsync(&dptr, N, _stream));
  return (void*)dptr;
}
extern "C" void mpMemcpyHtoD(void* dev_ptr, const void* cpu_ptr, len_t N, StreamCtx stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyHtoDAsync(dptr, cpu_ptr, N, _stream));
}
extern "C" void mpMemcpyDtoH(void* cpu_ptr, void const* dev_ptr, len_t N, StreamCtx stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyDtoHAsync(cpu_ptr, dptr, N, _stream));
}
extern "C" void mpMemFree(void* dev_ptr, StreamCtx stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipFreeAsync(dptr, _stream));
}
extern "C" void mpStreamSynchronize(StreamCtx stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  CURESULT_ASSERT(hipStreamSynchronize(_stream));
}
extern "C" void mpDeviceSynchronize() {
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" StreamCtx mpInitStream() {
  hipStream_t stream_ptr = nullptr;
  CURESULT_ASSERT(hipStreamCreateWithFlags(&stream_ptr, hipStreamDefault));
  return { .ptr = reinterpret_cast<void*>(stream_ptr) };
}
extern "C" void mpDeinitStream(StreamCtx stream) {
  hipStream_t _stream = static_cast<hipStream_t>(stream.ptr);
  CURESULT_ASSERT(hipStreamDestroy(_stream));
}
extern "C" void mpInitDevice(uint32_t device_number) {

    CURESULT_ASSERT(hipInit(device_number));

    hipDevice_t device;
    hipCtx_t context;
    int device_count = 0;

    CURESULT_ASSERT(hipGetDeviceCount(&device_count));

    if (device_count <= device_number) {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    CURESULT_ASSERT(hipDeviceGet(&device, device_number));

    CURESULT_ASSERT(hipCtxCreate(&context, 0, device));
}

// Convenience wrapper for hipGetLastError.
// TODO: make this return values instead of void
extern "C" void mpCheckLastError()
{
  CUDA_ASSERT(hipDeviceSynchronize());
  auto err = hipGetLastError();
  if (err != hipSuccess) {
      fprintf(stderr, "Error %s: %s", hipGetErrorName(err), hipGetErrorString(err));
  }
}

extern "C" len_t mpDeviceTotalMemory(uint32_t device) {
  len_t total;
  CURESULT_ASSERT(hipDeviceTotalMem(&total, device));
  return total;
}
