
#include "device_utils.h"

extern "C" void* mpMemAlloc(len_t N, StreamContext stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr;
  CURESULT_ASSERT(hipMallocAsync(&dptr, N, _stream));
  return (void*)dptr;
}
extern "C" void mpMemcpyHtoD(void* dev_ptr, const void* cpu_ptr, len_t N, StreamContext stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyHtoDAsync(dptr, cpu_ptr, N, _stream));
}
extern "C" void mpMemcpyDtoH(void* cpu_ptr, void const* dev_ptr, len_t N, StreamContext stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyDtoHAsync(cpu_ptr, dptr, N, _stream));
}
extern "C" void mpMemFree(void* dev_ptr, StreamContext stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipFreeAsync(dptr, _stream));
}
extern "C" void mpStreamSynchronize(StreamContext stream) {
  hipStream_t _stream = get_stream(stream);
  CURESULT_ASSERT(hipStreamSynchronize(_stream));
}
extern "C" void mpDeviceSynchronize() {
  CUDA_ASSERT(hipDeviceSynchronize());
}
extern "C" void mpInitDevice(uint32_t device_number) {

    CURESULT_ASSERT(hipInit(device_number));

    hipDevice_t device;
    hipCtx_t context;
    int device_count = 0;

    CURESULT_ASSERT(hipGetDeviceCount(&device_count));

    if (device_count <= device_number) {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    // get first CUDA device
    CURESULT_ASSERT(hipDeviceGet(&device, device_number));

    CURESULT_ASSERT(hipCtxCreate(&context, 0, device));
}

// Convenience wrapper for hipGetLastError.
// TODO: make this return values instead of void
extern "C" void mpCheckLastError()
{
  CUDA_ASSERT(hipDeviceSynchronize());
  auto err = hipGetLastError();
  if (err != hipSuccess) {
      fprintf(stderr, "Error %s: %s", hipGetErrorName(err), hipGetErrorString(err));
  }
}

extern "C" len_t mpDeviceTotalMemory(uint32_t device) {
  len_t total;
  CURESULT_ASSERT(hipDeviceTotalMem(&total, device));
  return total;
}

extern "C" StreamContext mpInitStream() {
  hipStream_t hip_stream = nullptr;
  hipblasHandle_t blas_handle = nullptr;

  // TODO: Add device parameter? This can set devices for creating
  //       the streams before initializing them.
  //          ex: hipSetDevice()

  CURESULT_ASSERT(hipStreamCreateWithFlags(&hip_stream, hipStreamDefault));

  CUBLAS_ASSERT(hipblasCreate(&blas_handle));

  CUBLAS_ASSERT(hipblasSetStream(blas_handle, hip_stream));

  return { 
    .hip_stream = { .ptr = reinterpret_cast<void*>(hip_stream) },
    .blas_handle = { .ptr = reinterpret_cast<void*>(blas_handle) }
  };
}

extern "C" void mpDeinitStream(StreamContext stream) {

  // TODO: If devices get set, it's probably a good idea to capture
  //       which device a stream was created on and put that in the
  //       StreamContext object. Research if it's required to deinit
  //       streams on the correct device.
  
  CUBLAS_ASSERT(hipblasDestroy(get_handle(stream)));
  CURESULT_ASSERT(hipStreamDestroy(get_stream(stream)));
}
