#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_cce_loss_ij_j_RScalar(
    const RScalar* src_value, 
          RScalar* src_grads, 
    const len_t* trgs,
          RScalar* scratch,
          double* redux, // scalar
    len_t m,
    len_t n
) {
      auto grid = cg::this_grid();
      
      // find our tile row and column
      len_t t_row = threadIdx.y;
      len_t t_col = threadIdx.x;

      const len_t row_offset = n * ((blockIdx.y * WARP_SIZE) + t_row);
      src_value += row_offset;
      src_grads += row_offset;

      // each row of m is handled by one thread warp (m / 32)
      const len_t m_pos = (blockIdx.y * WARP_SIZE) + t_row;
      const len_t trg = (m_pos < m) ? trgs[m_pos] : 0;

      if (t_col == 0)
            printf("(%d, %d): %d\n", (int)t_row, (int)t_col, (int)trg);

  //////////////////////////////////
  /// Grid Max /////////////////////

      RScalar grid_sum = RScalar(0.0f);

      for (len_t step = 0; step < n; step += WARP_SIZE) {

            if ((m_pos < m) && ((step + t_col) < n)) {    
                  const RScalar x = src_value[t_col];    
                  grid_sum += ((step + t_col) == trg) ? -rlog(MaxOP::apply(x, Init<RScalar>::epsilon())) : RScalar(0.0f);
                  // calculate the dydx
                  if (src_value != src_grads) {
                      src_grads[t_col] = x - RScalar(((step + t_col) == trg) ? 1.0f : 0.0f);
                  }
            }
            src_value += WARP_SIZE;
            src_grads += WARP_SIZE;
        }

      // no sum to be further calculated
      if (redux == nullptr)
            return;

      grid_sum = blockReduce<AddOP, WARP_SIZE>(
            grid_sum, t_row, t_col
      );

      if ((t_row == 0) && (t_col == 0)) {
            scratch[blockIdx.y] = grid_sum;
      }

      grid.sync();

      grid_sum = RScalar(0.0f);

      // now we change our thinking to make the block do the reduction
      if ((blockIdx.y == 0) && (m_pos < m)) {

            // flatten the matrix indexing
            const len_t idx = (t_row * WARP_SIZE) + t_col;

            for (len_t step = 0; step < gridDim.y; step += (WARP_SIZE * WARP_SIZE)) {

                  if ((step + idx) < gridDim.y) {
                        grid_sum += scratch[step + idx];
                  }
            }
      }
      __syncthreads();

      grid_sum = blockReduce<AddOP, WARP_SIZE>(
            grid_sum, t_row, t_col
      );

      //////////////////////////////////
      /// Grid Sum /////////////////////

      if ((blockIdx.y == 0) && (t_row == 0) && (t_col == 0)) {
            const double denom = static_cast<double>(m);
            *redux = static_cast<double>(grid_sum) / denom;
      }
} 

extern "C" void launch_cce_loss_ij_j_RScalar(
  StreamCtx stream,
  const RScalar* src_value, 
        RScalar* src_grads, 
  const len_t* trgs,
        RScalar* scratch,
        double* redux, // scalar
  len_t m,
  len_t n
) {

      dim3 grid (
        1, DIMPAD(m, WARP_SIZE)
      );

      dim3 block (
        WARP_SIZE, 
        WARP_SIZE
      );

      void* args[] = { 
        (void*)&src_value, 
        (void*)&src_grads, 
        (void*)&trgs, 
        (void*)&scratch, 
        (void*)&redux, 
        (void*)&m,
        (void*)&n
      };

      CUDA_ASSERT(hipLaunchCooperativeKernel(
        (void*)(__kernel_cce_loss_ij_j_RScalar), grid, block, args, 0, getCtx(stream)
      ));
}
