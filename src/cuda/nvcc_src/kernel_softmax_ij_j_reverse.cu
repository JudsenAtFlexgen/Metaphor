#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_softmax_ij_j_reverse_RScalar(
          RScalar* A_grads,
    const RScalar* B_value, 
    const RScalar* B_grads, 
    len_t m,
    len_t n
) {
  __shared__ RScalar redux[WARP_SIZE];

  // find our tile row and column
  len_t t_row = threadIdx.y;
  len_t t_col = threadIdx.x;

  const len_t m_bound = blockIdx.y * blockDim.y + t_row;

  A_grads += m_bound * n;
  B_value += m_bound * n;
  B_grads += m_bound * n;

  //////////////////////////////////
  /// Grid Sum /////////////////////

  auto B_val = B_value;
  auto B_grd = B_grads;

  RScalar grid_sum = RScalar(0.0f);

  for (len_t step = 0; step < m; step += blockDim.x) {

    if ((m_bound < m) && (step + t_col) < n) {    
      grid_sum += B_val[t_col] * B_grd[t_col];
    }
    // move A along the columns
    B_val += blockDim.x;
    B_grd += blockDim.x;
  }

  grid_sum = warpReduce<AddOP>(grid_sum);

  if (m_bound < m && t_col == 0) {
    redux[t_row] = grid_sum;
  }

  __syncthreads();

  if (m_bound < m) {
    grid_sum = redux[t_row];
  }
  
  for (len_t step = 0; step < n; step += blockDim.x) {

    if ((m_bound < m) && (step + t_col) < n) {    
      A_grads[t_col] = B_value[t_col] * (B_grads[t_col] - grid_sum);
    }
    // move A along the columns
    A_grads += blockDim.x;
    B_value += blockDim.x;
    B_grads += blockDim.x;
  }
} 

extern "C" void launch_softmax_ij_j_reverse_RScalar(
  StreamCtx stream,
        RScalar* A_grads,
  const RScalar* B_value, 
  const RScalar* B_grads,
  len_t m,
  len_t n
) {

  dim3 grid (
    1, DIMPAD(m, WARP_SIZE)
  );

  dim3 block (
    WARP_SIZE, 
    WARP_SIZE
  );

  __kernel_softmax_ij_j_reverse_RScalar<<<grid, block, 0, getCtx(stream)>>>(
    A_grads, B_value, B_grads, m, n
  );
}
