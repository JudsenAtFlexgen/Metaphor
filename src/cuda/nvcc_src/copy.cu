#include "hip/hip_runtime.h"
#include "../kernel_header.h"

// CUDA Kernel for Vector fill
__global__ void __kernel_copy_Scalar(
  const Scalar* src,
        Scalar* dst,
  unsigned n
) {
  // TODO: Implement block limiting
  
  // each thread loads 4 elements in each block
  const unsigned chunk_size = (blockDim.x * 4);

  // find our starting position
  const unsigned chunk_offset = chunk_size * blockIdx.x;

  // find thread position within chunk
  const unsigned pos = chunk_offset + (threadIdx.x * 4);

  if (pos < n) {
    // check if we have enough room for a coalesced load
    if (4 < (n - pos)) {
      auto src_cls = *reinterpret_cast<coalesce<Scalar>::c_ptr>(&src[pos]);
      auto dst_cls =  reinterpret_cast<coalesce<Scalar>::ptr>(&dst[pos]);
      *dst_cls = src_cls;
    }
    else {
      for (unsigned i = pos; i < n; ++i) dst[i] = src[i];
    }
  }
}

extern "C" void launch_copy_Scalar(
  const void* src, 
        void* dst, 
  len_t n,
  StreamContext stream
) {
  // TODO: search for hyper parameters
  dim3 grid_block(DIMPAD(n, (1024 * 4)), 1);
  dim3 thread_block(1024);
  __kernel_copy_Scalar<<<grid_block, thread_block, 0, get_stream(stream)>>>(
    static_cast<const Scalar*>(src), 
    static_cast<Scalar*>(dst), 
    static_cast<unsigned>(n)
  );
}


