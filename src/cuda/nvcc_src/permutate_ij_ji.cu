#include "hip/hip_runtime.h"
#include "../kernel_header.h"

#ifndef BLOCK_ROWS
#define BLOCK_ROWS 8
#endif

__global__ void __kernel_permutate_ij_ji(
    const Scalar* src,
          Scalar* dst,
    Scalar alpha, // used in reverse
    unsigned row,
    unsigned col
){
    __shared__ Scalar tile[WARP_SIZE][WARP_SIZE + 1];
    len_t i_row = blockIdx.y * WARP_SIZE + threadIdx.y; // <- threadIdx.y in [0, 7)
    len_t i_col = blockIdx.x * WARP_SIZE + threadIdx.x; // <- threadIdx.x in [0, 32)

    // scale the matrix in tiles, 4 times for each thread,
    // note that the tiles are smaller than the block size
    // by a factor of 4

    for (len_t i = 0; i < WARP_SIZE; i += BLOCK_ROWS){
        if(i_col < col  && (i_row + i) < row){
            tile[threadIdx.y + i][threadIdx.x] = src[(i_row + i) * col + i_col];
        }
    }
    __syncthreads();

    i_row = blockIdx.x * WARP_SIZE + threadIdx.y;
    i_col = blockIdx.y * WARP_SIZE + threadIdx.x; 

    for (len_t i = 0; i < WARP_SIZE; i += BLOCK_ROWS){
        if(i_col < row  && (i_row + i) < col){
            
            const len_t i_dst = (i_row + i) * row + i_col;

            // blend output with destination forward dst_coef is 0, reverse is 1
            dst[i_dst] = tile[threadIdx.x][threadIdx.y + i] + (alpha * dst[i_dst]);
        }
    }
}

extern "C" void launch_permutate_ij_ji_Scalar(
    const void* src,
          void* dst,
    double alpha,
    len_t row,
    len_t col,
    StreamContext ctx
) {
    const dim3 grid(
        DIMPAD(col, WARP_SIZE), 
        DIMPAD(row, WARP_SIZE)
    );

    const dim3 tile(
        WARP_SIZE, BLOCK_ROWS 
    );

    __kernel_permutate_ij_ji<<<grid, tile, 0, get_stream(ctx)>>>(
        static_cast<const Scalar*>(src), 
        static_cast<Scalar*>(dst),
        static_cast<Scalar>(alpha), 
        static_cast<unsigned>(row),
        static_cast<unsigned>(col)
    );
}
