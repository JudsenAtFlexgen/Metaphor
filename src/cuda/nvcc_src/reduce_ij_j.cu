#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_reduce_ij_j(
    const Scalar* src,
          Scalar* dst,
          Scalar alpha,
    len_t m,
    len_t n
){
    __shared__ Scalar smem[WARP_SIZE][WARP_SIZE + 1];
  
    // this version covers the n-dimension with blocks
    const unsigned n_pos = blockIdx.x * blockDim.x + threadIdx.x;

    // move to our starting row and column
    src += threadIdx.y * n + (blockDim.x * blockIdx.x);
    dst += (blockDim.x * blockIdx.x);

    Scalar col_sum = 0.0;

    for (unsigned m_step = 0; m_step < m; m_step += blockDim.y) {

      // TODO: create transpose boundary conditions to reduce smem reads
      smem[threadIdx.y][threadIdx.x] = 0.0f;

      const unsigned m_pos = m_step + threadIdx.y;

      if ((m_pos < m) && (n_pos < n)) {
        // transpose our load in
        smem[threadIdx.y][threadIdx.x] = src[threadIdx.x];
      }

      src += (blockDim.y * n);

      __syncthreads();

      col_sum += warpReduce<AddOP>(smem[threadIdx.x][threadIdx.y]);

      __syncthreads();
    }

    if (threadIdx.x == 0 && (threadIdx.y + blockDim.x * blockIdx.x) < n) {
      dst[threadIdx.y] = col_sum + alpha * dst[threadIdx.x];
    }

}

extern "C" void launch_reduce_ij_j_Scalar(
    const void* src,
          void* dst,
    double alpha,
    len_t m,
    len_t n,
    StreamContext stream
) {
    const dim3 grid(DIMPAD(m, WARP_SIZE));
    const dim3 block(WARP_SIZE, WARP_SIZE);

    __kernel_reduce_ij_j<<<grid, block, 0, get_stream(stream)>>>(
        static_cast<const Scalar*>(src), 
        static_cast<Scalar*>(dst),
        static_cast<Scalar>(alpha),
        static_cast<unsigned>(m),
        static_cast<unsigned>(n)
    );
}
