#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_selu_RScalar(
  const RScalar *a,
        RScalar *b,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < N) {
    const RScalar value = a[tid];    
    b[tid] = (value >= RScalar(0.0f)) ? value : rexp(value) - RScalar(1.0f);
  }
}

extern "C" void launch_selu_RScalar(
  StreamCtx stream,
  const RScalar* a,
        RScalar* b, 
  len_t N
) {
  __kernel_selu_RScalar<<<GRID_1D(N), dim3(1024), 0, getCtx(stream)>>>(a, b, N);
}
