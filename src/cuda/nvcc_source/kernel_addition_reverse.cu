#include "hip/hip_runtime.h"
#include "../kernel_header.h"

__global__ void __kernel_addition_reverse_RScalar(
  RScalar *dev_a,
  const RScalar *dev_b,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N)
    dev_a[tid] += dev_b[tid];
}

extern "C" void launch_addition_reverse_RScalar(
  StreamCtx stream,
  RScalar* a, 
  const RScalar* b, 
  len_t N
) {
  __kernel_addition_reverse_RScalar<<<GRID_1D(N), dim3(1024), 0, getCtx(stream)>>>(a, b, N);
}

__global__ void __kernel_addition_reverse_CScalar(
  CScalar *dev_a,
  const CScalar *dev_b,
  len_t N
) {
  const len_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
     
  if (tid < N) {
    dev_a[tid].r += dev_b[tid].r;
    dev_a[tid].i += dev_b[tid].i;
  }
}

extern "C" void launch_addition_reverse_CScalar(
  StreamCtx stream,
  CScalar* a, 
  const CScalar* b, 
  len_t N
) {
  __kernel_addition_reverse_CScalar<<<GRID_1D(N), dim3(1024), 0, getCtx(stream)>>>(a, b, N);
}
